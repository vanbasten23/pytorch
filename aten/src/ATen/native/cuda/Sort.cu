#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/native/cuda/Sort.h>
#include <ATen/core/TensorBase.h>
#include <ATen/core/Array.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/cub.cuh>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/KernelUtils.h>
#include <ATen/cuda/detail/OffsetCalculator.cuh>
#include <ATen/native/cuda/SortUtils.cuh>
#include <ATen/native/cuda/SortingCommon.cuh>

#include <limits>
#include <c10/core/DeviceArray.h>

namespace at { namespace native {

template <typename T>
static int minimum_grid_for_occupancy(T kernel, int max_block_size) {
  int minGridSize;
  int blockSize;
  C10_CUDA_CHECK(hipOccupancyMaxPotentialBlockSize(
      &minGridSize,
      &blockSize,
      kernel,
      /*dynamicSMemSize=*/0,
      max_block_size));
  return minGridSize;
}

// For very small sorts, use bitonicSortKVInPlace which performs
// better because it can sort multiple arrays within the same block of
// threads, improving occupancy.
//
// TODO: cub in CUDA 11.6 has a WarpMergeSort primitive that could
// replace the bitonic sort here.
struct SmallBitonicSort {
  template <int A, typename K, typename V, typename IndexType>
  void sort(
      at::cuda::detail::TensorInfo<K, IndexType> keyInfo,
      IndexType keySlices,
      IndexType keySliceSize,
      IndexType keySliceStride,
      at::cuda::detail::TensorInfo<V, IndexType> valueInfo,
      IndexType valueSliceStride,
      bool descending) {
    constexpr int sort_size = 32;
    constexpr int max_block_y = 16;
    constexpr int items_per_thread = 2;
    static_assert(sort_size % items_per_thread == 0, "");
    constexpr int block_x = sort_size / items_per_thread;

    TORCH_INTERNAL_ASSERT(keySliceSize <= sort_size);

    // Scale batch size down if the grid would be too small
    const auto min_grid = minimum_grid_for_occupancy(
        bitonicSortKVInPlace<
            A, -1, block_x, max_block_y,
            K, V, LTOp<K, true>, IndexType>,
        block_x * max_block_y);
    const auto max_batch = std::max(IndexType{1}, keySlices / min_grid);
    const int block_y = std::min(IndexType(max_block_y), max_batch);
    dim3 block(block_x, block_y);

    dim3 grid;
    const int grid_count = (keySlices + block_y - 1) / block_y;
    TORCH_INTERNAL_ASSERT(getGridFromTiles(grid_count, grid),
                          "Too many slices to sort");
    const auto stream = at::cuda::getCurrentCUDAStream();

    if (descending) {
      bitonicSortKVInPlace<A, -1, block_x, max_block_y>
        <<<grid, block, 0, stream>>>(
          keyInfo,
          keySlices,
          keySliceSize,
          keySliceStride,
          valueInfo,
          valueSliceStride,
          GTOp<K, true>());
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    } else {
      bitonicSortKVInPlace<A, -1, block_x, max_block_y>
        <<<grid, block, 0, stream>>>(
          keyInfo,
          keySlices,
          keySliceSize,
          keySliceStride,
          valueInfo,
          valueSliceStride,
          LTOp<K, true>());
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
  }
};

// For medium sizes (32 < n <= 4096) use radixSortKVInplace for better
// performance than the bitonic sort kernel.
struct MediumRadixSort {

  template <int A, typename K, typename V, typename IndexType>
  void sort(
      at::cuda::detail::TensorInfo<K, IndexType> keyInfo,
      IndexType keySlices,
      IndexType keySliceSize,
      IndexType keySliceStride,
      at::cuda::detail::TensorInfo<V, IndexType> valueInfo,
      IndexType valueSliceStride,
      bool descending) {

#define HANDLE_CASE(SIZE, ITEMS_PER_THREAD)         \
    fixed_size_sort<A, SIZE, ITEMS_PER_THREAD>(     \
        keyInfo,                                    \
        keySlices,                                  \
        keySliceSize,                               \
        keySliceStride,                             \
        valueInfo,                                  \
        valueSliceStride,                           \
        descending)

    int64_t ceilPowerOf2 = nextHighestPowerOf2(keySliceSize);
    TORCH_INTERNAL_ASSERT(ceilPowerOf2 <= 4096);
    switch (ceilPowerOf2) {
      case 4096:
        HANDLE_CASE(4096, 32);
        break;
      case 2048:
        HANDLE_CASE(2048, 32);
        break;
      case 1024:
      case 512:
      case 256:
        HANDLE_CASE(1024, 32);
        break;
      case 128:
      case 64:
        HANDLE_CASE(128, 4);
        break;
      case 32:
      case 16:
      case 8:
      case 4:
      case 2:
        HANDLE_CASE(32, 2);
        break;
      case 1:
        /* Nothing to do, data already sorted */
        break;
      default:
        TORCH_INTERNAL_ASSERT(false);
    }
#undef HANDLE_CASE

  }

  template <int A, int sort_size, int items_per_thread,
            typename K, typename V, typename IndexType>
  void fixed_size_sort(
      at::cuda::detail::TensorInfo<K, IndexType> keyInfo,
      IndexType keySlices,
      IndexType keySliceSize,
      IndexType keySliceStride,
      at::cuda::detail::TensorInfo<V, IndexType> valueInfo,
      IndexType valueSliceStride,
      bool descending) {
    static_assert(sort_size % items_per_thread == 0, "");
    constexpr int block = sort_size / items_per_thread;
    dim3 grid;
    TORCH_INTERNAL_ASSERT(getGridFromTiles(keySlices, grid),
                          "Too many slices to sort");

    const auto stream = at::cuda::getCurrentCUDAStream();
    radixSortKVInPlace<A, -1, block, items_per_thread>
        <<<grid, block, 0, stream>>>(
          keyInfo,
          keySlices,
          keySliceSize,
          keySliceStride,
          valueInfo,
          valueSliceStride,
          descending);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  }
};

template <typename Sorter>
void sortCommon(Sorter sorter, const TensorBase &key, const TensorBase &value,
                int dim, bool descending) {
  TORCH_CHECK(key.sizes() == value.sizes(),
              "Key tensor must have same size as value tensor");
  int dims = value.dim();
  TORCH_CHECK(dims <= MAX_DIMS, "value tensor has too many dimensions");
  // if key and value tensors have the same size, we do not need to check both

  ptrdiff_t inElements = key.numel();

  if (inElements == 0) {
    return;
  }

  int64_t keySliceSize = key.size(dim);
  ptrdiff_t keySlices = inElements / keySliceSize;

#define HANDLE_SORT_CASE(TYPE, A)                   \
  sorter.template sort<A>(                          \
      keyInfo,                                      \
      (TYPE) keySlices,                             \
      (TYPE) keySliceSize,                          \
      (TYPE) keyInfo.strides[collapseKeyDim],       \
      valueInfo,                                    \
      (TYPE) valueInfo.strides[collapseValueDim],   \
      descending)

  // The constructed key/value tensor info is used to select the slice
  // we are sorting on a per-block basis
  // The constructed key/value tensor info is used to select the slice
  // we are sorting on a per-block basis
  AT_DISPATCH_ALL_TYPES_AND3(at::ScalarType::Half, at::ScalarType::BFloat16, at::ScalarType::Bool, key.scalar_type(), "sortKeyValueInplace", [&]  {
    if (at::cuda::detail::canUse32BitIndexMath(key)) {
      at::cuda::detail::TensorInfo<scalar_t, unsigned int> keyInfo =
        at::cuda::detail::getTensorInfo<scalar_t, unsigned int>(key);
      at::cuda::detail::TensorInfo<int64_t, unsigned int> valueInfo =
        at::cuda::detail::getTensorInfo<int64_t, unsigned int>(value);

      auto strideKey = keyInfo.strides[dim];
      keyInfo.sizes[dim] = 1;
      int collapseKeyDim = keyInfo.collapseDims(dim);
      keyInfo.strides[collapseKeyDim] = strideKey;
      auto strideValue = valueInfo.strides[dim];
      valueInfo.sizes[dim]=1;
      int collapseValueDim = valueInfo.collapseDims(dim);
      valueInfo.strides[collapseValueDim] = strideValue;

      if (keyInfo.isContiguous()) {
        HANDLE_SORT_CASE(unsigned int, -2);
      } else {
        switch (keyInfo.dims) {
          case 2:
            HANDLE_SORT_CASE(unsigned int, 2);
            break;
          default:
            HANDLE_SORT_CASE(unsigned int, -1);
            break;
        }
      }

    } else {
      at::cuda::detail::TensorInfo<scalar_t, uint64_t> keyInfo =
        at::cuda::detail::getTensorInfo<scalar_t, uint64_t>(key);
      at::cuda::detail::TensorInfo<int64_t, uint64_t> valueInfo =
        at::cuda::detail::getTensorInfo<int64_t, uint64_t>(value);

      auto strideKey = keyInfo.strides[dim];
      keyInfo.sizes[dim] = 1;
      int collapseKeyDim = keyInfo.collapseDims(dim);
      keyInfo.strides[collapseKeyDim] = strideKey;
      auto strideValue = valueInfo.strides[dim];
      valueInfo.sizes[dim]=1;
      int collapseValueDim = valueInfo.collapseDims(dim);
      valueInfo.strides[collapseValueDim] = strideValue;

      // int64_t case is rare, just instantiate the generic version
      HANDLE_SORT_CASE(uint64_t, -1);
    }
  });
#undef HANDLE_SORT_CASE
}

void sortKeyValueInplace(
    const TensorBase& key,
    const TensorBase& value,
    int dim,
    bool descending,
    bool stable) {
  if (!stable && key.size(dim) <= 32) {
    // NOTE: Bitonic sort is unstable
    sortCommon(SmallBitonicSort{}, key, value, dim, descending);
  } else {
    sortCommon(MediumRadixSort{}, key, value, dim, descending);
  }
}

namespace {

struct offset_t {
  int stride;
  int begin;
  __device__ int operator[](int i) {
    return stride * (begin + i);
  }
};

}

namespace {

// Segmented sort by full sort algorithm:.
// Say we are sorting a (2, 3) tensor. We have in flattened form:
// values       0.4 1.2 5.3 6.2 1.3 2.3
// indices        0   1   2   0   1   2
// segment_id     0   0   0   1   1   1

// First we sort by values, globally:
// values       6.2 5.3 2.3 1.2 1.3 0.4
// indices        0   2   2   1   1   0
// segment_id     1   0   1   0   1   0

// Then we stable sort by segment id:
// values       5.3 1.2 0.4 6.2 2.3 1.3
// indices        2   1   0   0   2   1
// segment_id     0   0   0   1   1   1

// This method can only work if the slice we are sorting (`dim`) is
// innermost, and both values and indices are contiguous. We do this
// by re-arranging the input into this form as needed, which will
// unfortunately allocate memory if the request is not in this form.
// Vectorized sort is slower than iterated sort if the number of
// slices is small (since we're sorting twice, instead of invoking a
// smaller sort `numSlices` times), but the cub sort
// implementation here is a catch-all, so we're not looking for
// efficiency, but instead correctness.

template<typename scalar_t>
__global__ void sort_postprocess_kernel(const scalar_t *in, scalar_t *out, int64_t *index, const int2 *i_s_ptr, int nsegments, int nsort) {
  CUDA_KERNEL_LOOP(i, nsegments * nsort) {
    int segment = i / nsort;
    int j = i % nsort;

    int offset = segment * nsort;
    const scalar_t *in_ = in + offset;
    scalar_t *out_ = out + offset;
    int64_t *index_ = index + offset;
    const int2 *i_s_ptr_ = i_s_ptr + offset;

    int idx = i_s_ptr_[j].y;
    index_[j] = idx;
    out_[j] = in_[idx];
  }
}


C10_LAUNCH_BOUNDS_1(at::cuda::detail::CUDA_NUM_THREADS)
__global__ void fill_index_and_segment_kernel(
    int2 *data, int numel, at::cuda::detail::IntDivider<uint32_t> nsort_divider) {
  CUDA_KERNEL_LOOP(idx, numel) {
    auto div_mod = nsort_divider.divmod(idx);
    auto segment = static_cast<int>(div_mod.div);
    auto sort = static_cast<int>(div_mod.mod);
    data[idx] = int2{segment, sort};
  }
}

C10_LAUNCH_BOUNDS_1(at::cuda::detail::CUDA_NUM_THREADS)
__global__ void fill_reverse_indices_kernel(
    int64_t *data, int numel, at::cuda::detail::IntDivider<uint32_t> nsort_divider) {
  CUDA_KERNEL_LOOP(idx, numel) {
    data[idx] = nsort_divider.mod(idx);
  }
}

template<typename scalar_t>
inline void segmented_sort_large_segments(
    const int64_t nsegments, const int64_t nsort, const int64_t n, const bool descending,
    const scalar_t * self_ptr, scalar_t * values_ptr, int64_t * indices_ptr
  ) {
  using namespace at::cuda::detail;
  auto allocator = at::cuda::getCUDADeviceAllocator();
  auto stream = at::cuda::getCurrentCUDAStream();
  dim3 block = CUDA_NUM_THREADS;
  dim3 grid = GET_BLOCKS(nsort);
  c10::DeviceArray<int64_t> indices(*allocator, nsort);
  at::cuda::detail::IntDivider<uint32_t> nsort_divider(nsort);
  fill_reverse_indices_kernel<<<grid, block, 0, stream>>>(
      indices.get(), nsort, nsort_divider);
  const int64_t *initial_indices = indices.get();

  for (auto i: c10::irange(nsegments)){
    at::cuda::hipcub::radix_sort_pairs<scalar_t, int64_t>(
        self_ptr, values_ptr, initial_indices, indices_ptr,
        nsort, descending);
    indices_ptr += nsort;
    self_ptr += nsort;
    values_ptr += nsort;
  }
}

template<typename scalar_t>
inline void segmented_sort_pairs_by_full_sort(
  const int64_t nsegments, const int64_t nsort, const int64_t n, const bool descending,
  const scalar_t *const self_ptr, scalar_t *const values_ptr, int64_t *const indices_ptr
) {
  int64_t segment_bits = std::max<int64_t>(1L, static_cast<int64_t>(std::ceil(std::log2(nsegments))));

  const auto numel = nsort * nsegments;
  auto cuda_allocator = at::cuda::getCUDADeviceAllocator();
  auto indices_and_segment = cuda_allocator->allocate(numel * sizeof(int2));
  auto i_s_ptr = static_cast<int2 *>(indices_and_segment.get());

  using namespace at::cuda::detail;
  dim3 block = CUDA_NUM_THREADS;
  dim3 grid = GET_BLOCKS(numel);
  auto stream = c10::cuda::getCurrentCUDAStream();
  at::cuda::detail::IntDivider<uint32_t> nsort_divider(nsort);
  fill_index_and_segment_kernel<<<grid, block, 0, stream>>>(
      i_s_ptr, numel, nsort_divider);

  auto indices_and_segment2 = cuda_allocator->allocate(nsegments * nsort * sizeof(int2));
  auto i_s_ptr2 = static_cast<int2 *>(indices_and_segment2.get());

  at::cuda::hipcub::radix_sort_pairs<scalar_t, int2>(
    self_ptr, nullptr, i_s_ptr, i_s_ptr2,
    n, descending);

  TORCH_INTERNAL_ASSERT(segment_bits <= 32);

  // sort on lower 32bits, i.e. segment index
  at::cuda::hipcub::radix_sort_keys<int64_t>(
    reinterpret_cast<int64_t *>(i_s_ptr2), reinterpret_cast<int64_t *>(i_s_ptr),
    n, false, 0, segment_bits);

  sort_postprocess_kernel<<<(n + 511) / 512, 512, 0, at::cuda::getCurrentCUDAStream()>>>(
    self_ptr, values_ptr, indices_ptr, i_s_ptr, nsegments, nsort);
}

template<typename scalar_t>
void segmented_sort_pairs(
    int64_t nsegments, int64_t nsort, int64_t n, bool descending,
    const scalar_t *self_ptr, scalar_t *values_ptr, int64_t *indices_ptr) {
  const auto numel = nsort * nsegments;
  auto cuda_allocator = at::cuda::getCUDADeviceAllocator();
  auto reverse_indices = cuda_allocator->allocate(numel * sizeof(int64_t));
  int64_t *reverse_indices_ptr = static_cast<int64_t *>(reverse_indices.get());

  using namespace at::cuda::detail;
  dim3 block = CUDA_NUM_THREADS;
  dim3 grid = GET_BLOCKS(numel);
  auto stream = c10::cuda::getCurrentCUDAStream();
  at::cuda::detail::IntDivider<uint32_t> nsort_divider(nsort);
  fill_reverse_indices_kernel<<<grid, block, 0, stream>>>(
      reverse_indices_ptr, numel, nsort_divider);

  at::cuda::hipcub::segmented_sort_pairs(self_ptr, values_ptr,
                                      reverse_indices_ptr, indices_ptr, n, nsegments,
                                      offset_t{(int)nsort, 0}, offset_t{(int)nsort, 1}, descending);
}

}  // namespace

void launch_stable_sort_kernel(
    const TensorBase &self, int64_t dim, bool descending,
    const TensorBase &values, const TensorBase &indices) {
  const auto numel = self.numel();
  if (numel == 0) {
    return;
  }

  int64_t numel_or_intmax = std::min(numel, static_cast<int64_t>(std::numeric_limits<int>::max()));
  int64_t nsort = self.size(dim);
  int64_t nbatch = (numel_or_intmax / nsort) * nsort;
  TORCH_CHECK(nbatch > 0, "Cannot sort dimension of length ", nsort);
  int64_t *indices_ptr = indices.data_ptr<int64_t>();

#if (defined(USE_ROCM) && ROCM_VERSION < 40500)
  constexpr bool is_rocm_bf16_sort_unsupported = true;
#else
  constexpr bool is_rocm_bf16_sort_unsupported = false;
#endif

  AT_DISPATCH_ALL_TYPES_AND3(kBool, kHalf, kBFloat16, self.scalar_type(), "sort", [&]{
    c10::guts::if_constexpr<!(is_rocm_bf16_sort_unsupported && std::is_same<scalar_t, c10::BFloat16>::value)>([&](auto _){
      const scalar_t *self_ptr = self.data_ptr<scalar_t>();
      scalar_t *values_ptr = values.data_ptr<scalar_t>();
      int64_t remaining = _(numel);
      while (remaining > 0) {
        int64_t n = std::min(remaining, nbatch);
        int64_t nsegments = n / nsort;

        if (nsegments == 1 || nsort >= 1000000) { //rough heuristics where even a single sort occupies GPU
          segmented_sort_large_segments(
              nsegments, nsort, n, descending,
              self_ptr, values_ptr, indices_ptr);
        } else if (nsegments < 128) {
          segmented_sort_pairs_by_full_sort(nsegments, nsort, n, descending,
            self_ptr, values_ptr, indices_ptr);
        } else {
          segmented_sort_pairs(nsegments, nsort, n, descending,
                               self_ptr, values_ptr, indices_ptr);
        }

        remaining -= n;
        self_ptr += n;
        values_ptr += n;
        indices_ptr += n;
      }
    }, [&](auto _){ TORCH_CHECK(_(false), "BFloat16 is not supported on ROCm < 4.5"); });
  });
}

}}  // namespace at::native
